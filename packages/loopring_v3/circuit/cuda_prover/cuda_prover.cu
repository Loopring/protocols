#include <string>
#include <chrono>

#define NDEBUG 1

#include <prover_reference_functions.hpp>

#include "multiexp/reduce.cu"

extern void run_preprocess(const char *params_path, const char *preprocess_path);
// This is where all the FFTs happen

// template over the bundle of types and functions.
// Overwrites ca!
template <typename B>
typename B::vector_Fr *compute_H(size_t d, typename B::vector_Fr *ca,
                                 typename B::vector_Fr *cb,
                                 typename B::vector_Fr *cc) {
  auto domain = B::get_evaluation_domain(d + 1 + 1);

  B::domain_iFFT(domain, ca);
  B::domain_iFFT(domain, cb);

  size_t m = B::domain_get_m(domain);
  typename B::vector_Fr *coefficients_for_H = B::vector_Fr_zeros(m + 1);

  /* add coefficients of the polynomial (d2*A + d1*B - d3) + d1*d2*Z */
  B::domain_mul_add_sub(coefficients_for_H, ca, cb, m);
  B::domain_add_poly_Z(domain, coefficients_for_H);

  B::domain_cosetFFT(domain, ca);
  B::domain_cosetFFT(domain, cb);

  // Use ca to store H
  auto H_tmp = ca;

  // for i in 0 to m: H_tmp[i] = ca[i] * cb[i]
  B::vector_Fr_muleq(H_tmp, ca, cb, m);

  B::domain_iFFT(domain, cc);
  B::domain_cosetFFT(domain, cc);

  // for i in 0 to m: H_tmp[i] -= cc[i]
  B::vector_Fr_subeq(H_tmp, cc, m);

  B::domain_divide_by_Z_on_coset(domain, H_tmp);

  B::domain_icosetFFT(domain, H_tmp);

  // coefficients_for_H[i] += H_tmp[i];
  B::vector_Fr_add(coefficients_for_H, coefficients_for_H, H_tmp, m);

  return coefficients_for_H;
}

static size_t read_size_t(FILE* input) {
  size_t n;
  size_t readSize = fread((void *) &n, sizeof(size_t), 1, input);
  if (readSize != 1) {
    fprintf(stderr, "fread error");
    abort();
  }
  return n;
}

template< typename B >
struct ec_type;

template<>
struct ec_type<alt_bn128_libsnark> {
    typedef ECp_ALT_BN128 ECp;
    typedef ECp2_ALT_BN128 ECpe;
};


void
check_trailing(FILE *f, const char *name) {
    long bytes_remaining = 0;
    while (fgetc(f) != EOF)
        ++bytes_remaining;
    if (bytes_remaining > 0)
        fprintf(stderr, "!! Trailing characters in \"%s\": %ld\n", name, bytes_remaining);
}


static inline auto now() -> decltype(std::chrono::high_resolution_clock::now()) {
    return std::chrono::high_resolution_clock::now();
}

template<typename T>
void
print_time(T &t1, const char *str) {
    auto t2 = std::chrono::high_resolution_clock::now();
    auto tim = std::chrono::duration_cast<std::chrono::milliseconds>(t2 - t1).count();
    printf("%s: %ld ms\n", str, tim);
    t1 = t2;
}

template <typename B>
void run_prover(
        const char *params_path,
        const char *input_path,
        const char *output_path,
        const char *preprocessed_path)
{
    B::init_public_params();

    size_t primary_input_size = 1;

    auto beginning = now();
    auto t = beginning;

    FILE *params_file = fopen(params_path, "r");
    size_t d = read_size_t(params_file);
    size_t orig_d = read_size_t(params_file);
    size_t m = read_size_t(params_file);
    rewind(params_file);

    printf("d = %zu, orig_d = %zu, m = %zu\n", d, orig_d, m);

    typedef typename ec_type<B>::ECp ECp;
    typedef typename ec_type<B>::ECpe ECpe;

    typedef typename B::G1 G1;
    typedef typename B::G2 G2;

    static constexpr int R = 32;
    static constexpr int C = 4;
    FILE *preprocessed_file = fopen(preprocessed_path, "r");

    size_t space = ((m + 1) + R - 1) / R;
    size_t space_H = ((d) + R - 1) / R;

    auto A_mults = load_points_affine<ECp>(((1U << C) - 1)*(m + 1), preprocessed_file);
    auto out_A = allocate_memory(space * ECp::NELTS * ELT_BYTES);

    auto B1_mults = load_points_affine<ECp>(((1U << C) - 1)*(m + 1), preprocessed_file);
    auto out_B1 = allocate_memory(space * ECp::NELTS * ELT_BYTES);

    auto B2_mults = load_points_affine<ECpe>(((1U << C) - 1)*(m + 1), preprocessed_file);
    auto out_B2 = allocate_memory(space * ECpe::NELTS * ELT_BYTES);

    auto L_mults = load_points_affine<ECp>(((1U << C) - 1)*(m - 1), preprocessed_file);
    auto out_L = allocate_memory(space * ECp::NELTS * ELT_BYTES);

    auto H_mults = load_points_affine<ECp>(((1U << C) - 1)*(d), preprocessed_file);
    auto out_H = allocate_memory(space_H * ECp::NELTS * ELT_BYTES);

    fclose(preprocessed_file);

    print_time(t, "load preprocessing");

    auto params = B::read_params(params_file, d, m);
    fclose(params_file);
    print_time(t, "load params");

    auto t_main = t;

    FILE *inputs_file = fopen(input_path, "r");
    auto w_ = load_scalars(m + 1, inputs_file);
    rewind(inputs_file);
    auto inputs = B::read_input(inputs_file, d, orig_d + 1, m);
    fclose(inputs_file);
    print_time(t, "load inputs");

    const var *w = w_.get();

    auto t_gpu = t;

    hipStream_t sA, sB1, sB2, sL, sH;

    ec_reduce_straus<ECp, C, R>(sA, out_A.get(), A_mults.get(), w, m + 1);
    ec_reduce_straus<ECp, C, R>(sB1, out_B1.get(), B1_mults.get(), w, m + 1);
    ec_reduce_straus<ECpe, C, 2*R>(sB2, out_B2.get(), B2_mults.get(), w, m + 1);
    ec_reduce_straus<ECp, C, R>(sL, out_L.get(), L_mults.get(), w + (primary_input_size + 1) * ELT_LIMBS, m - 1);
    print_time(t, "gpu launch");

    //G1 *evaluation_At = B::multiexp_G1(B::input_w(inputs), B::params_A(params), m + 1);
    //G1 *evaluation_Bt1 = B::multiexp_G1(B::input_w(inputs), B::params_B1(params), m + 1);
    //G2 *evaluation_Bt2 = B::multiexp_G2(B::input_w(inputs), B::params_B2(params), m + 1);

    // Do calculations relating to H on CPU after having set the GPU in
    // motion
    auto coefficients_for_H =
        compute_H<B>(orig_d, B::input_ca(inputs), B::input_cb(inputs), B::input_cc(inputs));
    print_time(t, "coefficients_for_H");

    auto H_coeff_mem = allocate_memory(d * ELT_BYTES);
    B::coefficients_for_H_to_mem(coefficients_for_H, (uint8_t *)H_coeff_mem.get(), ELT_BYTES, d);
    print_time(t, "coefficients_H_mem");
#if 0
    auto H = B::params_H(params);
    G1 *evaluation_Ht = B::multiexp_G1(coefficients_for_H, H, d);
    B::delete_vector_G1(H);
    print_time(t, "evaluation_Ht");
#else
    ec_reduce_straus<ECp, C, R>(sH, out_H.get(), H_mults.get(), H_coeff_mem.get(), d);
#endif

    print_time(t, "cpu 1");

    hipDeviceSynchronize();
    hipStreamSynchronize(sA);
    G1 *alpha_g1 = B::alpha_g1(params);
    G1 *evaluation_At = B::read_pt_ECp(out_A.get());
    auto final_At = B::G1_add(alpha_g1, evaluation_At);

    hipStreamSynchronize(sB1);
    G1 *evaluation_Bt1 = B::read_pt_ECp(out_B1.get());
    auto final_Bt1 = B::G1_add(B::beta_g1(params), evaluation_Bt1);

    hipStreamSynchronize(sB2);
    G2 *evaluation_Bt2 = B::read_pt_ECpe(out_B2.get());
    auto final_Bt2 = B::G2_add(B::beta_g2(params), evaluation_Bt2);

    hipStreamSynchronize(sL);
    G1 *evaluation_Lt = B::read_pt_ECp(out_L.get());

    hipStreamSynchronize(sH);
    G1 *evaluation_Ht = B::read_pt_ECp(out_H.get());

    print_time(t_gpu, "gpu e2e");


    auto scaled_Bt1 = B::G1_scale(B::input_r(inputs), final_Bt1);
    auto Lt1_plus_scaled_Bt1 = B::G1_add(evaluation_Lt, scaled_Bt1);
    auto final_C = B::G1_add(evaluation_Ht, evaluation_Lt);

    print_time(t, "cpu 2");

    B::groth16_output_write(final_At, final_Bt2, final_C, inputs, output_path);

    print_time(t, "store");

    print_time(t_main, "Total time from input to output: ");

    hipStreamDestroy(sA);
    hipStreamDestroy(sB1);
    hipStreamDestroy(sB2);
    hipStreamDestroy(sL);
    hipStreamDestroy(sH);

    B::delete_G1(evaluation_At);
    B::delete_G1(evaluation_Bt1);
    B::delete_G2(evaluation_Bt2);
    B::delete_G1(evaluation_Ht);
    B::delete_G1(evaluation_Lt);
    B::delete_G1(scaled_Bt1);
    B::delete_G1(Lt1_plus_scaled_Bt1);
    B::delete_vector_Fr(coefficients_for_H);
    B::delete_groth16_input(inputs);
    B::delete_groth16_params(params);

    print_time(t, "cleanup");
    print_time(beginning, "Total runtime (incl. file reads)");
}

int main(int argc, char **argv) {
  printf("main start\n");
  setbuf(stdout, NULL);
  std::string mode(argv[1]);

  const char *params_path = argv[2];

  if (mode == "compute") {
      const char *input_path = argv[3];
      const char *preprocess_path = argv[4];
      const char *output_path = argv[5];
      run_prover<alt_bn128_libsnark>(params_path, input_path, output_path, preprocess_path);
  } else if (mode == "preprocess") {
        const char *preprocess_path = argv[3];
        run_preprocess(params_path, preprocess_path);
  }

  return 0;
}
