#include "hip/hip_runtime.h"
#pragma once

#include "arith.cu"

// All algorithms from
// https://www.hyperelliptic.org/EFD/g1p/auto-shortw-jacobian.html#addition-add-2007-bl
template< typename FF, int CRV_A, typename Grp >
struct ec_jac {
    typedef FF field_type;

    // NB: This is corresponds to the group of rational points for
    // curves over prime field; it is a lie for curves over extension
    // fields.
    typedef Grp group_type;

    // TODO: See if using Chudnovsky coordinates improves things much
    // (ie. store Z^2 and Z^3 as well)
    FF x, y, z;

    static constexpr int NELTS = 3 * FF::DEGREE; // *3 for x, y and z

    __device__
    static void
    load_affine(ec_jac &P, const var *mem) {
        FF::load(P.x, mem);
        FF::load(P.y, mem + FF::DEGREE * ELT_LIMBS);
        FF::set_one(P.z);

        // FIXME: This is an odd convention, but that's how they do it.
        if (FF::is_zero(P.x))
            set_zero(P);
    }

    __device__
    static void
    load_jac(ec_jac &P, const var *mem) {
        FF::load(P.x, mem);
        FF::load(P.y, mem + FF::DEGREE * ELT_LIMBS);
        FF::load(P.z, mem + 2 * FF::DEGREE * ELT_LIMBS);
    }

    __device__
    static int
    is_affine(const ec_jac &P) {
        FF one;
        FF::set_one(one);
        return FF::are_equal(P.z, one);
    }

    __device__
    static int
    are_equal(const ec_jac &P, const ec_jac &Q) {
        FF zPzP, zPzPzP, zQzQ, zQzQzQ;

        FF::sqr(zPzP, P.z);
        FF::sqr(zQzQ, Q.z);

        FF t0, t1;
        FF::mul(t0, P.x, zQzQ);
        FF::mul(t1, Q.x, zPzP);

        if ( ! FF::are_equal(t0, t1))
            return 0;

        // x-coordinates are equal; now check the y-coordinates.

        FF::mul(zPzPzP, zPzP, P.z);
        FF::mul(zQzQzQ, zQzQ, Q.z);
        FF::mul(t0, P.y, zQzQzQ);
        FF::mul(t1, Q.y, zPzPzP);

        return FF::are_equal(t0, t1);
    }

#if 0
    __device__
    static void
    store_affine(var *mem, const ec_jac &P) {
        FF z_inv, z2_inv, z3_inv, aff_x, aff_y;

        // NB: Very expensive!
        // TODO: Consider (i) doing this on the host and (ii) implementing
        // simultaneous inversion.
        FF::inv(z inv, P.z);
        FF::sqr(z2_inv, z_inv);
        FF::mul(z3_inv, z2_inv, z_inv);

        FF::mul(aff_x, P.x, z2_inv);
        FF::store(mem, aff_x);

        FF::mul(aff_y, P.y, z3_inv);
        FF::store(mem + FF::DEGREE * ELT_LIMBS, aff_y);
    }
#endif

    __device__
    static void
    store_jac(int t, var *mem, const ec_jac &P) {
#if 0
        printf("t %d store_jac\n", t);
        printf("t %d P.x %llx\n", t, P.x);
        printf("t %d P.y %llx\n", t, P.y);
        printf("t %d P.z %llx\n", t, P.z);
#endif
        FF::store(mem, P.x);
        FF::store(mem + FF::DEGREE * ELT_LIMBS, P.y);
        FF::store(mem + 2 * FF::DEGREE * ELT_LIMBS, P.z);
    }

    __device__
    static void
    set_zero(ec_jac &P) {
#if 0
        FF::set_one(P.x);
        FF::set_one(P.y);
        FF::set_zero(P.z);
#else
        FF::set_zero(P.x);
        FF::set_one(P.y);
        FF::set_zero(P.z);
#endif
    }

    __device__
    static int
    is_zero(const ec_jac &P) { return FF::is_zero(P.z); }

#if 0
    // TODO: Needs double-checking
    __device__
    static void
    mixed_dbl(ec_jac &R, const ec_jac &P) {
        FF xx, yy, yyyy, s, m, t, t0, t1;

        FF::sqr(xx, P.x);      // XX = X1^2
        FF::sqr(yy, P.y);      // YY = Y1^2
        FF::sqr(yyyy, yy);     // YYYY = YY^2
        FF::add(s, P.x, yy);   // t0 = X1+YY
        FF::sqr(s, s);         // t1 = t0^2
        FF::sub(s, s, xx);     // t2 = t1-XX
        FF::sub(s, s, yyyy);   // t3 = t2-YYYY
        mul_<2>::x(s);         // S = 2*t3
        mul_<3>::x(m, xx);     // t4 = 3*XX

        // FIXME: Won't work
        FF::add(m, m, CRV_A);  // M = t4+a

        FF::sqr(t, m);         // t5 = M^2
        mul_<2>::x(t0, s);     // t6 = 2*S
        FF::sub(t, t, t0);     // T = t5-t6
        R.x = t;               // X3 = T
        mul_<2>::x(R.z, P.y);  // Z3 = 2*Y1
        FF::sub(t0, s, t);     // t7 = S-T
        mul_<8>::x(t1, yyyy);  // t8 = 8*YYYY
        FF::mul(R.y, m, t0);   // t9 = M*t7
        FF::sub(R.y, R.y, t1); // Y3 = t9-t8
    }
#endif

    __device__
    static void
    mixed_add(int t, ec_jac &R, const ec_jac &P, const ec_jac &Q) {
#if 0
        printf("t %d mixed add\n", t);
        printf("t %d P.x a0 %llx\n", t, P.x.a0);
        printf("t %d P.x a1 %llx\n", t, P.x.a1);
        printf("t %d P.y a0 %llx\n", t, P.y.a0);
        printf("t %d P.y a1 %llx\n", t, P.y.a1);
        printf("t %d P.z a0 %llx\n", t, P.z.a0);
        printf("t %d P.z a1 %llx\n", t, P.z.a1);
        printf("t %d Q.x a0 %llx\n", t, Q.x.a0);
        printf("t %d Q.x a1 %llx\n", t, Q.x.a1);
        printf("t %d Q.y a0 %llx\n", t, Q.y.a0);
        printf("t %d Q.y a1 %llx\n", t, Q.y.a1);
        printf("t %d Q.z a0 %llx\n", t, Q.z.a0);
        printf("t %d Q.z a1 %llx\n", t, Q.z.a1);
#endif
        // Would be better to know that Q != 0
        if (is_zero(Q)) {
            R = P;
            return;
        } else if (is_zero(P)) {
            R = Q;
            return;
        }
        assert(is_affine(Q));

        FF t0, t1;
#if 0
        FF z1z1, u2, s2, h, hh, i, j, r, v;

        FF::sqr(z1z1, P.z);     // Z1Z1 = Z1^2
        FF::mul(u2, Q.x, z1z1); // U2 = X2*Z1Z1
        FF::mul(s2, Q.y, P.z);
        FF::mul(s2, s2, z1z1);  // S2 = Y2*Z1*Z1Z1
        if (FF::are_equal(u2, P.x) && FF::are_equal(s2, P.y)) {
            // P == Q
            //mixed_dbl(R, Q);
            dbl(R, Q);
            return;
        }
        FF::sub(h, u2, P.x);    // H = U2-X1
        FF::sqr(hh, h);         // HH = H^2
        mul_<4>::x(i, hh);      // I = 4*HH
        FF::mul(j, h, i);       // J = H*I
        FF::sub(r, s2, P.y);    // t1 = S2-Y1
        mul_<2>::x(r, r);       // r = 2*t1
        FF::mul(v, P.x, i);     // V = X1*I

        FF::sqr(t0, r);         // t2 = r^2
        mul_<2>::x(t1, v);      // t3 = 2*V
        FF::sub(t0, t0, j);     // t4 = t2-J
        FF::sub(R.x, t0, t1);   // X3 = t4-t3

        FF::sub(t0, v, R.x);    // t5 = V-X3
        FF::mul(t1, P.y, j);    // t6 = Y1*J
        mul_<2>::x(t1, t1);     // t7 = 2*t6
        FF::mul(t0, r, t0);     // t8 = r*t5
        FF::sub(R.y, t0, t1);   // Y3 = t8-t7

        FF::add(t0, P.z, h);    // t9 = Z1+H
        FF::sqr(t0, t0);        // t10 = t9^2
        FF::sub(t0, t0, z1z1);  // t11 = t10-Z1Z1
        FF::sub(R.z, t0, hh);   // Z3 = t11-HH
#endif
#if 1
        FF Z1Z1, U2, Z1_cubed, S2;
        FF::sqr(Z1Z1, P.z);
        FF::mul(U2, Q.x, Z1Z1);
        FF::mul(Z1_cubed, P.z, Z1Z1);
        FF::mul(S2, Q.y, Z1_cubed);

        if (FF::are_equal(P.x, U2) && FF::are_equal(P.y, S2))
        {
            dbl(t, R, Q);
            return;
        }

        FF H, HH, I, J, r, V;


        FF::sub(H, U2, P.x);
        FF::sqr(HH, H);
        mul_<4>::x(I, HH);

        FF::mul(J, H, I);
        FF::sub(t0, S2, P.y);
        mul_<2>::x(r, t0);
        FF::mul(V, P.x, I);

        FF::sqr(t0, r);
        FF::sub(t1, t0, J);
        FF::sub(t0, t1, V);
        FF::sub(R.x, t0, V);

        FF t2;
        FF::sub(t0, V, R.x);
        FF::mul(t1, r, t0);
        FF::mul(t0, P.y, J);
        mul_<2>::x(t2, t0);
        FF::sub(R.y, t1, t2);

        FF::add(t0, R.z, H);
        FF::sqr(t1, t0);
        FF::sub(t0, t1, Z1Z1);
        FF::sub(R.z, t0, HH);
#endif
#if 0
        printf("t %d mixed add result\n", t);
        printf("t %d R.x a0 %llx\n", t, R.x.a0);
        printf("t %d R.x a1 %llx\n", t, R.x.a1);
        printf("t %d R.y a0 %llx\n", t, R.y.a0);
        printf("t %d R.y a1 %llx\n", t, R.y.a1);
        printf("t %d R.z a0 %llx\n", t, R.z.a0);
        printf("t %d R.z a1 %llx\n", t, R.z.a1);
#endif
    }

    // NB: This is not valid if P = Q or if P == 0 or Q == 0
    __device__
    static void
    add_unsafe(int t, ec_jac &R, const ec_jac &P, const ec_jac &Q) {
#if 0
        printf("t %d add unsafe\n", t);
        printf("t %d P.x a0 %llx\n", t, P.x.a0);
        printf("t %d P.x a1 %llx\n", t, P.x.a1);
        printf("t %d P.y a0 %llx\n", t, P.y.a0);
        printf("t %d P.y a1 %llx\n", t, P.y.a1);
        printf("t %d P.z a0 %llx\n", t, P.z.a0);
        printf("t %d P.z a1 %llx\n", t, P.z.a1);
        printf("t %d Q.x a0 %llx\n", t, Q.x.a0);
        printf("t %d Q.x a1 %llx\n", t, Q.x.a1);
        printf("t %d Q.y a0 %llx\n", t, Q.y.a0);
        printf("t %d Q.y a1 %llx\n", t, Q.y.a1);
        printf("t %d Q.z a0 %llx\n", t, Q.z.a0);
        printf("t %d Q.z a1 %llx\n", t, Q.z.a1);
#endif
#if 0
        printf("t %d unsafe add\n", t);
        printf("t %d P.x %llx\n", t, P.x);
        printf("t %d P.y %llx\n", t, P.y);
        printf("t %d P.z %llx\n", t, P.z);
        printf("t %d Q.x %llx\n", t, Q.x);
        printf("t %d Q.y %llx\n", t, Q.y);
        printf("t %d Q.z %llx\n", t, Q.z);
#endif
        FF t0, t1;
#if 0
        FF z1z1, z2z2, u1, u2, s1, s2, h, i, j, r, v;

        FF::sqr(z1z1, P.z); // Z1Z1 = Z1^2
        FF::sqr(z2z2, Q.z); // Z2Z2 = Z2^2
        FF::mul(u1, P.x, z2z2); // U1 = X1*Z2Z2
        FF::mul(u2, Q.x, z1z1); // U2 = X2*Z1Z1
        FF::mul(s1, P.y, Q.z);
        FF::mul(s1, s1, z2z2); // S1 = Y1*Z2*Z2Z2
        FF::mul(s2, Q.y, P.z);
        FF::mul(s2, s2, z1z1); // S2 = Y2*Z1*Z1Z1
        FF::sub(h, u2, u1); // H = U2-U1
        mul_<2>::x(i, h);
        FF::sqr(i, i); // I = (2*H)^2
        FF::mul(j, h, i); // J = H*I
        FF::sub(r, s2, s1);
        mul_<2>::x(r, r); // r = 2*(S2-S1)
        FF::mul(v, u1, i); // V = U1*I

        // X3 = r^2-J-2*V
        FF::sqr(t0, r);
        FF::sub(t0, t0, j);
        mul_<2>::x(t1, v);
        FF::sub(R.x, t0, t1);

        // Y3 = r*(V-X3)-2*S1*J
        FF::sub(t0, v, R.x);
        FF::mul(t0, r, t0);
        FF::mul(t1, s1, j);
        mul_<2>::x(t1, t1);
        FF::sub(R.y, t0, t1);

        // Z3 = ((Z1+Z2)^2-Z1Z1-Z2Z2)*H
        FF::add(t0, P.z, Q.z);
        FF::sqr(t0, t0);
        FF::add(t1, z1z1, z2z2);
        FF::sub(t0, t0, t1);
        FF::mul(R.z, t0, h);
#else
        FF Z1Z1, Z2Z2, U1, U2, Z1_cubed, Z2_cubed, S1, S2;

        FF::sqr(Z1Z1, P.z);
        FF::sqr(Z2Z2, Q.z);

        FF::mul(U1, P.x, Z2Z2);
        FF::mul(U2, Q.x, Z1Z1);

        FF::mul(Z1_cubed, P.z, Z1Z1);
        FF::mul(Z2_cubed, Q.z, Z2Z2);

        FF::mul(S1, P.y, Z2_cubed);
        FF::mul(S2, Q.y, Z1_cubed);

        if (FF::are_equal(U1, U2) && FF::are_equal(S1, S2))
        {
            dbl(t, R, Q);
            return;
        }

        FF H, S2_minus_S1, I, J, r, V, S1_J;

        FF::sub(H, U2, U1);
        FF::sub(S2_minus_S1, S2, S1);
        FF::add(t0, H, H);
        FF::sqr(I, t0);
        FF::mul(J, H, I);
        FF::add(r, S2_minus_S1, S2_minus_S1);
        FF::mul(V, U1, I);

        FF::sqr(t0, r);
        FF::sub(t1, t0, J);
        FF::sub(t0, t1, V);
        FF::sub(R.x, t0, V);

        FF::mul(S1_J, S1, J);
        FF::sub(t0, V, R.x);
        FF::mul(t1, r, t0);
        mul_<2>::x(t0, S1_J);
        FF::sub(R.y, t1, t0);

        FF::add(t0, P.z, Q.z);
        FF::sqr(t1, t0);
        FF::sub(t0, t1, Z1Z1);
        FF::sub(t1, t0, Z2Z2);
        FF::mul(R.z, t1, H);
#if 0
        printf("t %d unsafe add result\n", t);
        printf("t %d R.x %llx\n", t, R.x);
        printf("t %d R.y %llx\n", t, R.y);
        printf("t %d R.z %llx\n", t, R.z);
#endif
#if 0
        printf("t %d unsafe add result\n", t);
        printf("t %d R.x a0 %llx\n", t, R.x.a0);
        printf("t %d R.x a1 %llx\n", t, R.x.a1);
        printf("t %d R.y a0 %llx\n", t, R.y.a0);
        printf("t %d R.y a1 %llx\n", t, R.y.a1);
        printf("t %d R.z a0 %llx\n", t, R.z.a0);
        printf("t %d R.z a1 %llx\n", t, R.z.a1);
#endif
#endif
    }

    __device__
    static void
    add(int t, ec_jac &R, const ec_jac &P, const ec_jac &Q) {
        // TODO: It should be the caller's responsibility to check if
        // the operands are zero
        // Need P != 0 and Q != 0 for computation below to work
        if (is_zero(P)) {
            R = Q;
            return;
        } else if (is_zero(Q)) {
            R = P;
            return;
        }

        // need to save P (or Q) just in case &R = &P and we need to
        // double P after the add.
        ec_jac PP = P;
        add_unsafe(t, R, P, Q);

        // If P = Q, then add returns all zeros.
        if (FF::is_zero(R.x) && FF::is_zero(R.y) && FF::is_zero(R.z)) {
            dbl(t, R, PP);
        }
    }

    __device__
    static void
    dbl(int t, ec_jac &R, const ec_jac &P) {
        FF t0, t1;

#if 0
        printf("t %d dbl\n", t);
        printf("t %d P.x a0 %llx\n", t, P.x.a0);
        printf("t %d P.x a1 %llx\n", t, P.x.a1);
        printf("t %d P.y a0 %llx\n", t, P.y.a0);
        printf("t %d P.y a1 %llx\n", t, P.y.a1);
        printf("t %d P.z a0 %llx\n", t, P.z.a0);
        printf("t %d P.z a1 %llx\n", t, P.z.a1);
#endif
#if 0
        printf("t %d dbl\n", t);
        printf("T %d P.x %llx\n", t, P.x);
        printf("T %d P.y %llx\n", t, P.y);
        printf("T %d P.z %llx\n", t, P.z);
#endif
#ifndef NDEBUG
        // TODO: It should be the caller's responsibility to check if
        // the operand is zero
        // Need P != 0 for computation below to work.
        if (is_zero(P)) {
            set_zero(R);
            return;
        }
#endif

#if 0
        FF xx, yy, yyyy, zz, s, m, t;
        FF::sqr(xx, P.x); // XX = X1^2
        FF::sqr(yy, P.y); // YY = Y1^2
        FF::sqr(yyyy, yy); // YYYY = YY^2
        FF::sqr(zz, P.z); // ZZ = Z1^2
        FF::add(t0, P.x, yy);
        FF::sqr(t0, t0);
        FF::add(t1, xx, yyyy);
        FF::sub(t0, t0, t1);
        mul_<2>::x(s, t0); // S = 2*((X1+YY)^2-XX-YYYY)
        mul_<3>::x(t0, xx);
        FF::sqr(t1, zz);
        mul_<CRV_A>::x(t1, t1);
        FF::add(m, t0, t1); // M = 3*XX+a*ZZ^2
        FF::sqr(t0, m);
        mul_<2>::x(t1, s);
        FF::sub(t, t0, t1); // T = M^2-2*S

        // X3 = T
        R.x = t;

        // NB: Need to do Z3 before Y3 in case &R = &P, since we need
        // to use P.y here.
        // Z3 = (Y1+Z1)^2-YY-ZZ
        FF::add(t0, P.y, P.z);
        FF::sqr(t0, t0);
        FF::add(t1, yy, zz);
        FF::sub(R.z, t0, t1);

        // Y3 = M*(S-T)-8*YYYY
        FF::sub(t0, s, t);
        FF::mul(t0, m, t0);
        mul_<8>::x(t1, yyyy);
        FF::sub(R.y, t0, t1);
#else
        FF A, B, C, D, E, F, eightC, Y1Z1;
        FF::sqr(A, P.x);
        FF::sqr(B, P.y);
        FF::sqr(C, B);

        FF::add(t0, P.x, B);
        FF::sqr(t1, t0);
        FF::sub(t0, t1, A);
        FF::sub(t1, t0, C);
        FF::add(D, t1, t1);

        mul_<3>::x(E, A);

        FF::sqr(F, E);

        FF::add(t0, D, D);
        FF::sub(R.x, F, t0);

        FF::mul(Y1Z1, P.y, P.z);
        FF::add(R.z, Y1Z1, Y1Z1);

        mul_<8>::x(eightC, C);
        FF::sub(t0, D, R.x);
        FF::mul(t1, E, t0);
        FF::sub(R.y, t1, eightC);

#endif
#if 0
        printf("t %d dbl result\n", t);
        printf("t %d R.x %llx\n", t, R.x);
        printf("t %d R.y %llx\n", t, R.y);
        printf("t %d R.z %llx\n", t, R.z);
#endif
#if 0
        printf("t %d dbl result\n", t);
        printf("t %d R.x a0 %llx\n", t, R.x.a0);
        printf("t %d R.x a1 %llx\n", t, R.x.a1);
        printf("t %d R.y a0 %llx\n", t, R.y.a0);
        printf("t %d R.y a1 %llx\n", t, R.y.a1);
        printf("t %d R.z a0 %llx\n", t, R.z.a0);
        printf("t %d R.z a1 %llx\n", t, R.z.a1);
#endif
    }

    template< int EXP >
    __device__ __forceinline__
    static void
    mul_2exp(int t, ec_jac &R, const ec_jac &P) {
        dbl(t, R, P);
        #pragma unroll
        for (int k = 1; k < EXP; ++k)
            dbl(t, R, R);
    }

    __device__
    static void
    neg(ec_jac &R, const ec_jac &P) {
        R.x = P.x;
        FF::neg(R.y, P.y);
        R.z = P.z;
    }

    __device__
    static void
    mul(ec_jac &R, const var &n, const ec_jac &P) {
        printf("mul n %d\n", n);
        // TODO: This version makes an effort to prevent intrawarp
        // divergence at a performance cost. This is probably no
        // longer a worthwhile trade-off.

        // TODO: Work out how to use add instead of add_safe.

        static constexpr int WINDOW_SIZE = 4;

        // TODO: I think it is better to use the remainder window
        // first rather than last. When it's last we sometimes miss
        // opportunities to use precomputed values.

        // Window decomposition: digit::BITS = q * WINDOW_SIZE + r.
        static constexpr unsigned WINDOW_REM_BITS = digit::BITS % WINDOW_SIZE;
        static constexpr unsigned WINDOW_MAX = (1U << WINDOW_SIZE);

        static constexpr unsigned WINDOW_MASK = (1U << WINDOW_SIZE) - 1U;
        static constexpr unsigned WINDOW_REM_MASK = (1U << WINDOW_REM_BITS) - 1U;

        if (is_zero(P)) {
            R = P;
            return;
        }

        /* G[t] = [t]P, t >= 0 */
        // TODO: This should be precomputed for all P.
        ec_jac G[WINDOW_MAX];
        set_zero(G[0]);
        G[1] = P;
        dbl(0, G[2], P);
        for (int t = 3; t < WINDOW_MAX; ++t)
            add(G[t], G[t - 1], P);

        auto g = fixnum::layout();

        int digit_idx = fixnum::most_sig_dig(n);
        if (digit_idx < 0) {
            // n == 0
            R = G[0];
            return;
        }

        // First iteration
        var f = g.shfl(n, digit_idx);

        // "Remainder"
        int j = digit::BITS - WINDOW_REM_BITS;
        var win = (f >> j) & WINDOW_REM_MASK;
        R = G[win];
        j -= WINDOW_SIZE;

        for (; j >= 0; j -= WINDOW_SIZE) {
            mul_2exp<WINDOW_SIZE>(0, R, R);
            win = (f >> j) & WINDOW_MASK;
            add(R, R, G[win]);
        }

        --digit_idx;
        for ( ; digit_idx >= 0; --digit_idx) {
            var f = g.shfl(n, digit_idx);
            var win; // TODO: Morally this should be an int

            // "Remainder"
            int j = digit::BITS - WINDOW_REM_BITS;
            mul_2exp<WINDOW_REM_BITS>(0, R, R);
            win = (f >> j) & WINDOW_REM_MASK;
            add(R, R, G[win]);

            j -= WINDOW_SIZE;

            for (; j >= 0; j -= WINDOW_SIZE) {
                mul_2exp<WINDOW_SIZE>(0, R, R);
                win = (f >> j) & WINDOW_MASK;
                add(R, R, G[win]);
            }
        }
    }
};



typedef ec_jac< Fp_ALT_BN128, 2, Fp_ALT_BN128_R > ECp_ALT_BN128;
typedef ec_jac< Fp2_ALT_BN128, 2*13, Fp_ALT_BN128_R > ECp2_ALT_BN128;
